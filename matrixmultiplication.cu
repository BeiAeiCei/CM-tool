#include "hip/hip_runtime.h"
#include<stdio.h>
#include"tool\common.cuh"
#include<math.h>
#include"tool\cudatime.cuh"
#include"tool\compare.cuh"
#include <time.h>
#include <windows.h>
// 矩阵乘实现
__global__
void CUDAmatrixMultiplication(float *A, float *B, float *C, int row_max, int line_max) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < row_max && col < line_max) {
        float sum = 0;
        for (int k = 0; k < row_max; k++) {
            sum += A[row * row_max + k] * B[k * line_max + col];
        }
        C[row * line_max + col] = sum;
    }
}
/**************************************************/

void BASEmatrixMultiplication(float *A, float *B, float *C, int row_max, int line_max, int MAX_num) {
    for (int row = 0; row < row_max; row++) {
        for (int col = 0; col < line_max; col++) {
            float sum = 0.0;
                for (int k = 0; k < line_max; k++) {
                    sum += A[row * line_max + k] * B[k * line_max + col];
                }
            C[row * line_max + col] = sum;
        }
    }
}

// 生成随机数，初始化ABC
__host__
void initdata(float *addr , int elemCount)
{
    for(int i = 0;i < elemCount;i++)
    {
        addr[i] = (float)(rand() & 0xff) / 10.f;
    }
}
/**************************************************/



int main()
{
    setGPU();
// 定义数据大小和矩阵大小
    int row_max;
    int line_max;
    printf("please write :row_max  line_max\n");
    scanf("%d %d",&row_max,&line_max);
    int MAX_num = row_max * line_max;
    int MAX_size = MAX_num * sizeof(float);
/**************************************************/


// 分配主机端内存
    float *A_H, *B_H , *C_H , *C_H1;
    A_H = (float *)malloc(MAX_size);
    B_H = (float *)malloc(MAX_size);
    C_H = (float *)malloc(MAX_size);
    C_H1 = (float *)malloc(MAX_size);
/**************************************************/


// 初始化主机端内存
    if(A_H != NULL && B_H != NULL && C_H != NULL){
        memset(A_H , 0 , MAX_size);
        memset(B_H , 0 , MAX_size);
        memset(C_H , 0 , MAX_size);
        memset(C_H1 , 0 , MAX_size);
    } else {
        printf("fail to allocate memory\n");
        free(A_H);
        free(B_H);
        free(C_H);
        free(C_H1);
        exit(-1);
    }
/**************************************************/

   srand(666);


// 初始化设备端内存
float *A_D, *B_D ,*C_D;
   hipMalloc((void **)&A_D,MAX_size);
   hipMalloc((void **)&B_D,MAX_size);
   hipMalloc((void **)&C_D,MAX_size);

    initdata(A_H , MAX_num);
    initdata(B_H , MAX_num);
/**************************************************/


// 数据搬运
    hipMemcpy(A_D,A_H,MAX_size,hipMemcpyHostToDevice);
    hipMemcpy(B_D,B_H,MAX_size,hipMemcpyHostToDevice);
    hipMemcpy(C_D,C_H,MAX_size,hipMemcpyHostToDevice);
/**************************************************/
float CUDAtotal_time = 0;
hipEvent_t start, stop; 

dim3 block(16, 16);
dim3 grid((line_max + block.x - 1) / block.x, (row_max + block.y - 1) / block.y);

    time_start(&start , &stop ,__FILE__ , __LINE__);

    for(int i = 0;i<11;i++){

    CUDAmatrixMultiplication<<<grid, block>>>(A_D, B_D, C_D, row_max, line_max);
    hipMemcpy(C_H, C_D, MAX_size, hipMemcpyDeviceToHost);

    float single_time;
    time_end(&start, &stop, &single_time, __FILE__, __LINE__);
    CUDAtotal_time += single_time;
    if(i!=10)memset(C_H, 0, MAX_size);
    }

    CUDAtotal_time /= 10;
    
Sleep(2000);
    clock_t start1, end1;
    double total_cpu_time_used = 0;
    for (int i = 0; i < 10; ++i) {
        start1 = clock();
        BASEmatrixMultiplication(A_H, B_H, C_H1, row_max, line_max, MAX_num);
        end1 = clock();
        total_cpu_time_used += ((double)(end1 - start1)) * 1000 / CLOCKS_PER_SEC;
        // 每次迭代后重置C矩阵，以便下一次迭代
        if(i!=9)memset(C_H1, 0, MAX_size);
    }
    double average_cpu_time_used = total_cpu_time_used / 10;
// 比较结果
    printf("-------------------------------------------------\n");
    if(compareMatrices(C_H , C_H1 , row_max , line_max , tolerance)){
        printf("\033[34mTest Success\n");
        float ratio = 0.0;
        ratio = average_cpu_time_used / CUDAtotal_time;
        printf("ratio is :%.7f\n\033[0m",ratio);
    }else{
        printf("\033[31mTest False\n\033[0m");
    }
    printf("-------------------------------------------------\n");
/**************************************************/


// 释放内存
    free(A_H);
    free(B_H);
    free(C_H);
    free(C_H1);
    hipFree(A_D);
    hipFree(B_D);
    hipFree(C_D);
/**************************************************/


return 0;
}